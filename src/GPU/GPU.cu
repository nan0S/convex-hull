#include "hip/hip_runtime.h"
#include "GPU.cuh"

#include <cstdio>
#include <iostream>

#include <GL/glew.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/zip_function.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>
#include <thrust/random.h>

#include "GPU/Error.h"
#include "Graphics/Error.h"
#include "Utils/Timer.h"
#include "Utils/Log.h"

namespace GPU
{
   /* macros */
   #define PI 3.14159265358f
   #define SEND_TO_GPU(symbol, expr) { \
               auto v = (expr); \
               cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(symbol), &v, sizeof(v), 0, \
                                           hipMemcpyHostToDevice)); }

   /* structs */
   struct generate_points
   {
      __device__
      void operator()(float& x, float& y) const;
   };

   struct is_above_line
   {
      __device__
      bool operator()(float x, float y) const;
   };

   struct calc_first_pts
   {
      __device__
      void operator()(int head, int key, int index) const;
   };

   struct calc_line_dist
   {
      __device__
      float operator()(float x, float y, int key, int hull_count) const;
   };

   struct update_heads
   {
      __device__
      void operator()(int index) const;
   };

   struct calc_outerior
   {
      __device__
      bool operator()(float x, float y, int key, int head, int hull_count) const;
   };

   struct is_on_hull
   {
      __device__
      bool operator()(int index, int hull_count) const;
   };

   struct GPUGenerator
   {
      hiprandGenerator_t gen;
      bool is_init;
   };

   struct CPUGenerator
   {
      thrust::minstd_rand rng;
      thrust::uniform_real_distribution<float> adist;
      thrust::uniform_real_distribution<float> rdist;
      bool is_init;
   };

   struct Memory
   {
      bool is_host_mem;
      GLuint gl_buffer;
      hipGraphicsResource_t resource;
      void* d_buffer;
      float* h_buffer;
   };

   /* forward declarations */
   __device__
   float cross(float ux, float uy, float vx, float vy);
   size_t getCudaMemoryNeeded(int n);

   /* constants */
   constexpr int CURAND_USAGE_THRESHOLD = 12'000'000;

   /* variables */
   GPUGenerator gpu_gen;
   CPUGenerator cpu_gen;
   Memory mem;

   __constant__ float d_r_min;
   __constant__ float d_r_max;

   __constant__ float* d_x;
   __constant__ float* d_y;
   __constant__ int* d_head;
   __constant__ int* d_first_pts;
   __constant__ int* d_flag;

   __constant__ float d_left_x;
   __constant__ float d_left_y;
   __constant__ float d_right_x;
   __constant__ float d_right_y;

   void init(Config config,
             const std::vector<int>& n_points,
             GLuint gl_buffer)
   {
      int max_n = -1, max_n_below_curand_threshold = -1;
      for (int n : n_points)
      {
         max_n = std::max(max_n, n);
         if (n < CURAND_USAGE_THRESHOLD)
            max_n_below_curand_threshold = std::max(
               max_n_below_curand_threshold, n);
      }

      float r_min = 0.f, r_max = 1.f;
      switch (config.dataset_type)
      {
         case DatasetType::DISC:
            r_min = 0.f; r_max = 1.f;
            break;
         case DatasetType::RING:
            r_min = 0.9f; r_max = 1.f;
            break;
         case DatasetType::CIRCLE:
            r_min = 1.f; r_max = 1.f;
            break;
         default:
            assert(false);
      }
      if (max_n_below_curand_threshold != -1)
      {
         cpu_gen.rng.seed(config.seed);
         cpu_gen.adist.param(decltype(cpu_gen.adist)::param_type(0, 2 * PI));
         cpu_gen.rdist.param(decltype(cpu_gen.rdist)::param_type(r_min, r_max));
         cpu_gen.is_init = true;
      }
      if (max_n >= CURAND_USAGE_THRESHOLD)
      {
         curandCall(hiprandCreateGenerator(&gpu_gen.gen, HIPRAND_RNG_PSEUDO_MT19937));
         curandCall(hiprandSetPseudoRandomGeneratorSeed(gpu_gen.gen, config.seed));
         gpu_gen.is_init = true;
         cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_r_min), &r_min, sizeof(float)));
         cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_r_max), &r_max, sizeof(float)));
      }

      mem.is_host_mem = config.is_host_mem;

      size_t cuda_needed = getCudaMemoryNeeded(max_n);
      if (!mem.is_host_mem)
      {
         // Allocate OpenGL buffer and prepare to map it into CUDA.
         glCall(glBufferData(GL_ARRAY_BUFFER, cuda_needed, NULL, GL_STATIC_DRAW));
         cudaCall(hipGraphicsGLRegisterBuffer(&mem.resource, gl_buffer,
                                               cudaGraphicsMapFlagsWriteDiscard));
         if (cpu_gen.is_init)
         {
            size_t host_bytes = 2 * max_n_below_curand_threshold * sizeof(float);
            cudaCall(hipHostMalloc(&mem.h_buffer, host_bytes));
         }
      }
      else
      {
         // Allocate OpenGL, CUDA and host buffers. CUDA->host->OpenGL->draw.
         size_t bytes = 2 * max_n * sizeof(float);
         glCall(glBufferData(GL_ARRAY_BUFFER, bytes, NULL, GL_STATIC_DRAW));
         cudaCall(hipMalloc(&mem.d_buffer, cuda_needed));
         cudaCall(hipHostMalloc(&mem.h_buffer, bytes));
      }

      glCall(glVertexAttribPointer(0, 1, GL_FLOAT, GL_FALSE, 0, 0));
   }

   int calculate(int n)
   {
      print("\nRunning GPU for ", n, " points.");

      // Initialize pointers to previously allocated memory.
      size_t cuda_needed = getCudaMemoryNeeded(n);
      if (!mem.is_host_mem)
      {
         size_t size = 0;
         cudaCall(hipGraphicsMapResources(1, &mem.resource));
         cudaCall(hipGraphicsResourceGetMappedPointer(&mem.d_buffer, &size, mem.resource));
         assert(size >= cuda_needed);
      }
      cudaCall(hipMemset(mem.d_buffer, 0, cuda_needed));

      thrust::device_ptr<float> x(reinterpret_cast<float*>(mem.d_buffer));
      thrust::device_ptr<float> y(reinterpret_cast<float*>(x.get() + n));
      thrust::device_ptr<int> head(reinterpret_cast<int*>(y.get() + n));
      thrust::device_ptr<int> keys(reinterpret_cast<int*>(head.get() + n));
      thrust::device_ptr<int> first_pts(reinterpret_cast<int*>(keys.get() + n));
      thrust::device_ptr<int> flag(reinterpret_cast<int*>(first_pts.get() + n));
      thrust::device_ptr<float> dist(reinterpret_cast<float*>(flag.get() + n));

      SEND_TO_GPU(d_x, x.get());
      SEND_TO_GPU(d_y, y.get());
      SEND_TO_GPU(d_head, head.get());
      SEND_TO_GPU(d_first_pts, first_pts.get());
      SEND_TO_GPU(d_flag, flag.get());

      // Generate points.
      if (n < CURAND_USAGE_THRESHOLD)
      {
         // Use CPU.
         float* h_x = mem.h_buffer, * h_y = h_x + n;
         for (int i = 0; i < n; ++i)
         {
            float r = cpu_gen.rdist(cpu_gen.rng);
            float a = cpu_gen.adist(cpu_gen.rng);
            *h_x++ = r * cos(a);
            *h_y++ = r * sin(a);
         }
         thrust::copy(mem.h_buffer, h_y, x);
      }
      else
      {
         // Use GPU (cuRAND).
         curandCall(hiprandGenerateUniform(gpu_gen.gen, x.get(), 2 * n));
         thrust::for_each_n(thrust::make_zip_iterator(x, y), n,
                            thrust::make_zip_function(generate_points{}));
      }

      Timer timer("QuickHull");

      // Find leftmost and rightmost points.
      auto it = thrust::minmax_element(thrust::make_zip_iterator(x, y),
                                       thrust::make_zip_iterator(x+n, y+n));
      auto it_left = it.first.get_iterator_tuple();
      auto it_right = it.second.get_iterator_tuple();
      cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_left_x), it_left.get<0>().get(),
                                  sizeof(float), 0, hipMemcpyDeviceToDevice));
      cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_left_y), it_left.get<1>().get(),
                                  sizeof(float), 0, hipMemcpyDeviceToDevice));
      cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_right_x), it_right.get<0>().get(),
                                  sizeof(float), 0, hipMemcpyDeviceToDevice));
      cudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_right_y), it_right.get<1>().get(),
                                  sizeof(float), 0, hipMemcpyDeviceToDevice));
      int left_idx = static_cast<int>(it_left.get<0>() - x);
      int right_idx = static_cast<int>(it_right.get<0>() - x);

      // Partition into lower and upper parts.
      auto pivot = thrust::partition(thrust::make_zip_iterator(x, y),
                                     thrust::make_zip_iterator(x+n, y+n),
                                     thrust::make_zip_function(is_above_line{}));
      int pivot_idx = static_cast<int>(pivot.get_iterator_tuple().get<0>() - x);

      // Sort points in lower and upper parts.
      thrust::sort(thrust::make_zip_iterator(x, y), pivot,
                   thrust::greater<>());
      thrust::sort(pivot, thrust::make_zip_iterator(x+n, y+n));

      // Initialize head.
      head[0] = 1;
      head[pivot.get_iterator_tuple().get<0>() - x] = 1;

      // Prepare variables.
      int hull_count = 0;
      int last_hull_count = 0;
      const int N = n;
      auto diter = thrust::make_discard_iterator();

      while (hull_count < n)
      {
         // Calculate keys from head.
         thrust::device_ptr<int> end = thrust::inclusive_scan(head, head+n, keys);
         hull_count = *(end - 1);
         // Line distance calculation ensured that segment borders will not
         // be selected as the farthest point in the segment (unless there
         // aren't anymore points in the segment). However if there still is
         // some precision-related issue, then this check is a guard from
         // an infinite loop. It should be always false, however I leave it
         // just in case (hull will be correct with respect to float::eps).
         if (hull_count == last_hull_count)
            break;
         last_hull_count = hull_count;
         thrust::for_each_n(keys, n, thrust::placeholders::_1 -= 1);

         // Calculate first_pts from keys and head.
         thrust::counting_iterator<int> iter(0);
         thrust::for_each(thrust::make_zip_iterator(head, keys, iter),
                          thrust::make_zip_iterator(head+n, keys+n, iter+n),
                          thrust::make_zip_function(calc_first_pts{}));

         // Calculate distances from segment lines.
         auto hull_count_citer = thrust::make_constant_iterator<int>(hull_count);
         thrust::transform(thrust::make_zip_iterator(x, y, keys, hull_count_citer),
                           thrust::make_zip_iterator(x+n, y+n, keys+n, hull_count_citer),
                           dist,
                           thrust::make_zip_function(calc_line_dist{}));

         // Find farthest points in segments.
         thrust::device_ptr<int> reduction_border =
            thrust::reduce_by_key(// reduction keys
                                  keys, keys+n,
                                  // values input
                                  thrust::make_zip_iterator(dist, thrust::make_counting_iterator(0)),
                                  // keys output - throw away
                                  diter,
                                  // values output - only care about index
                                  thrust::make_zip_iterator(diter, flag),
                                  // use maximum to reduce
                                  thrust::equal_to<>(), thrust::maximum<>())
            .second.get_iterator_tuple().get<1>();

         // Update heads with farthest points.
         thrust::for_each(flag, reduction_border, update_heads{});

         // Determine outerior points.
         auto outerior = thrust::device_ptr<int>(reinterpret_cast<int*>(dist.get()));
         thrust::transform(thrust::make_zip_iterator(x, y, keys, head, hull_count_citer),
                           thrust::make_zip_iterator(x+n, y+n, keys+n, head+n, hull_count_citer),
                           outerior,
                           thrust::make_zip_function(calc_outerior{}));

         // Discard interior points.
         n = static_cast<int>(
               thrust::stable_partition(thrust::make_zip_iterator(x, y, head),
                                        thrust::make_zip_iterator(x+n, y+n, head+n),
                                        outerior,
                                        // move outerior points to the beginning
                                        thrust::placeholders::_1 == 1)
               .get_iterator_tuple().get<0>() - x);
      }

      // Filter potentially at most one point that is one the line between
      // its neightbours.
      if (n > 2)
      {
         auto count_iter = thrust::make_counting_iterator(0);
         auto const_iter = thrust::make_constant_iterator(n);
         hull_count = static_cast<int>(
            thrust::stable_partition(thrust::make_zip_iterator(count_iter, const_iter),
                                     thrust::make_zip_iterator(count_iter+n, const_iter+n),
                                     thrust::make_zip_function(is_on_hull{}))
            .get_iterator_tuple().get<0>() - count_iter);
      }

      timer.stop();

      if (!mem.is_host_mem)
      {
         cudaCall(hipGraphicsUnmapResources(1, &mem.resource));
      }
      else
      {
         // Copy: CUDA->host->OpenGL.
         size_t bytes = 2 * N * sizeof(float);
         cudaCall(hipMemcpy(mem.h_buffer, mem.d_buffer, bytes, hipMemcpyDeviceToHost));
         glCall(glBufferSubData(GL_ARRAY_BUFFER, 0, bytes, mem.h_buffer));
      }

      glCall(glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0,
                                   reinterpret_cast<const void*>(N * sizeof(float))));

      return hull_count;
   }

   void cleanup()
   {
      if (!mem.is_host_mem)
      {
         cudaCall(hipGraphicsUnregisterResource(mem.resource));
         if (cpu_gen.is_init)
         {
            cudaCall(hipHostFree(mem.h_buffer));
         }
      }
      else
      {
         cudaCall(hipFree(mem.d_buffer));
         cudaCall(hipHostFree(mem.h_buffer));
      }

      if (gpu_gen.is_init)
      {
         curandCall(hiprandDestroyGenerator(gpu_gen.gen));
      }
   }

   __device__
   void generate_points::operator()(float& x, float& y) const
   {
      float a = x * 2 * PI;
      float r = (d_r_max - d_r_min) * y + d_r_min;
      x = r * cos(a);
      y = r * sin(a);
   }

   __device__
   bool is_above_line::operator()(float x, float y) const
   {
      if (x == d_right_x && y == d_right_y)
         return true;
      // Unfortunately it might happen that even though (x, y) is
      // leftmost point, still cross(...) > 0 (precision problems?).
      if (x == d_left_x && y == d_left_y)
         return false;
      float ux = x - d_right_x, uy = y - d_right_y;
      float vx = d_left_x - d_right_x, vy = d_left_y - d_right_y;
      return cross(ux, uy, vx, vy) > 0;
   }

   __device__
   void calc_first_pts::operator()(int head, int key, int index) const
   {
      if (head == 1)
         d_first_pts[key] = index;
   }

   __device__
   float calc_line_dist::operator()(float x, float y, int key, int hull_count)
      const
   {
      int nxt = key + 1;
      if (nxt == hull_count) nxt = 0;

      int i = d_first_pts[key];
      int j = d_first_pts[nxt];

      // Due to precision problems we have to explicitly ensure that
      // segmenent borders will not be selected as the farthest points in the
      // segment (points stricly inside segment might have distance 0 from
      // the segment line even though they are not on it) becuase it leads
      // to an infinite loop for the main algorithm.
      float x1 = d_x[i], y1 = d_y[i];
      if (x == x1 && y == y1)
         return -1.f;
      float x2 = d_x[j], y2 = d_y[j];
      if (x == x2 && y == y2)
         return -1.f;

      float dx = x2 - x1, dy = y2 - y1;
      float ux = x1 - x, uy = y1 - y;

      return cross(dx, dy, ux, uy);
   }

   __device__
   void update_heads::operator()(int index) const
   {
      d_head[index] = 1;
   }

   __device__
   bool calc_outerior::operator()(float x, float y, int key, int head,
      int hull_count) const
   {
      if (head) return true;

      int nxt = key + 1;
      if (nxt == hull_count) nxt = 0;

      int a = d_first_pts[key];
      int b = d_first_pts[nxt];
      int c = d_flag[key];

      float cx = d_x[c], cy = d_y[c];
      float ux = d_x[a] - cx, uy = d_y[a] - cy;
      x -= cx; y -= cy;
      if (cross(ux, uy, x, y) > 0)
         return true;

      float vx = d_x[b] - cx, vy = d_y[b] - cy;
      return cross(x, y, vx, vy) > 0;
   }

   __device__
   bool is_on_hull::operator()(int index, int hull_count) const
   {
      int prv = index - 1;
      if (prv == -1) prv = hull_count - 1;
      int nxt = index + 1;
      if (nxt == hull_count) nxt = 0;

      float px = d_x[prv], py = d_y[prv];
      float ux = d_x[index] - px, uy = d_y[index] - py;
      float vx = d_x[nxt] - px, vy = d_y[nxt] - py;

      return cross(ux, uy, vx, vy) != 0;
   }

   __device__
   float cross(float ux, float uy, float vx, float vy)
   {
      return ux * vy - vx * uy;
   }

   size_t getCudaMemoryNeeded(int n)
   {
      return n * (3 * sizeof(float) + 4 * sizeof(int));
   }

} // namespace GPU
